#include "hip/hip_runtime.h"
// #include "chap4.hpp"

#include "gpu.hpp"
#include <cstdlib>

#include <hip/hip_runtime.h>

// c4v0 //////////////////////////////////////////////////////////////

__global__ void kernel_c4v0(int n, const float *data, float *result)
{
    // 1. Convert block/thread coordinate to global coordinate
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= n)
        return;

    // 2. Start calculating values
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k)
    {
        v = min(v, data[n * i + k] + data[n * k + j]);
    }

    result[n * i + j] = v;
}

void GPU::c4v0(int n, const float *data, float *result)
{
    printf("Version: c4v0-own\n");

    int nBytes = n * n * sizeof(float);

    // 1. Allocate memory on GPU & copy date CPU -> GPU
    float *dGPU = nullptr, *rGPU = nullptr;
    hipMalloc((void **)&dGPU, nBytes);
    hipMalloc((void **)&rGPU, nBytes);

    hipMemcpy(dGPU, data, nBytes, hipMemcpyHostToDevice);

    // 2. Define block size & run kernel
    int nBlocks = (n + 16 - 1) / 16;
    dim3 dimThreads(16, 16);
    dim3 dimBlocks(nBlocks, nBlocks);

    kernel_c4v0<<<dimBlocks, dimThreads>>>(n, dGPU, rGPU);

    // 3. Copy data from GPU -> CPU
    hipMemcpy(result, rGPU, nBytes, hipMemcpyDeviceToHost);

    // 4. Release mem in GPU
    hipFree(dGPU);
    hipFree(rGPU);
}

// c4v1 //////////////////////////////////////////////////////////////

__global__ void kernel_c4v1(int n, const float *data, float *result)
{
    // 1. Convert block/thread coordinate to global coordinate
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= n)
        return;

    // 2. Start calculating values
    float v = HUGE_VALF;
    for (int k = 0; k < n; ++k)
    {
        v = min(v, data[n * j + k] + data[n * k + i]);
    }

    result[n * i + j] = v;
}

void GPU::c4v1(int n, const float *data, float *result)
{
    printf("Version: c4v1-own\n");

    int nBytes = n * n * sizeof(float);

    // 1. Allocate memory on GPU & copy date CPU -> GPU
    float *dGPU = nullptr, *rGPU = nullptr;
    hipMalloc((void **)&dGPU, nBytes);
    hipMalloc((void **)&rGPU, nBytes);

    hipMemcpy(dGPU, data, nBytes, hipMemcpyHostToDevice);

    // 2. Define block size & run kernel
    int nBlocks = (n + 16 - 1) / 16;
    dim3 dimThreads(16, 16);
    dim3 dimBlocks(nBlocks, nBlocks);

    kernel_c4v0<<<dimBlocks, dimThreads>>>(n, dGPU, rGPU);

    // 3. Copy data from GPU -> CPU
    hipMemcpy(result, rGPU, nBytes, hipMemcpyDeviceToHost);

    // 4. Release mem in GPU
    hipFree(dGPU);
    hipFree(rGPU);
}

// c4v2 //////////////////////////////////////

void mykernel(float *r, const float *d, int n, int nn)
{
    int tX = threadIdx.x;
    int tY = threadIdx.y;
    int bX = blockIdx.x;
    int bY = blockIdx.y;

    const float *t = d + nn * nn;

    float v[8][8];
    for (int i_ = 0; i_ < 8; ++i_)
    {
        for (int j_ = 0; j_ < 8; ++j_)
        {
            v[i_][j_] = HUGE_VALF;
        }
    }
    for (int k = 0; k < n; ++k)
    {
        float x[8];
        float y[8];
        for (int i_ = 0; i_ < 8; ++i_)
        {
            int i = bX * 64 + i_ * 8 + tX;
            x[i_] = t[nn * k + i];
        }
        for (int j_ = 0; j_ < 8; ++j_)
        {
            int j = bY * 64 + j_ * 8 + tY;
            y[j_] = d[nn * k + j];
        }
        for (int i_ = 0; i_ < 8; ++i_)
        {
            for (int j_ = 0; j_ < 8; ++j_)
            {
                v[i_][j_] = min(v[i_][j_], x[i_] + y[j_]);
            }
        }
    }
    for (int i_ = 0; i_ < 8; ++i_)
    {
        for (int j_ = 0; j_ < 8; ++j_)
        {
            int i = bX * 64 + i_ * 8 + tX;
            int j = bY * 64 + j_ * 8 + tY;
            if (i < n && j < n)
            {
                r[n * i + j] = v[i_][j_];
            }
        }
    }
}