
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <vector>

using namespace std;

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

__global__ void kernel(float *d, float *r, int ny, int nx)
{
    // Convert to global coordinate
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < i || i >= ny || j >= ny)
    {
        return;
    }
    else if (j == i)
    {
        r[ny * i + j] = 1;
    }
    else
    {
        float cor = 0;
        for (int k = 0; k < nx; ++k)
            cor += d[i * nx + k] * d[j * nx + k];

        r[i * ny + j] = cor;
    }
}

/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
void correlate(int ny, int nx, const float *data, float *result)
{
    float *norm = new float[ny * nx];

    // 1. row-wise 0-mean normalization

    // #pragma omp parallel for
    for (int i = 0; i < ny; ++i)
    {
        float mean = 0;
        for (int j = 0; j < nx; ++j)
            // #pragma omp critical
            mean += data[i * nx + j];

        mean /= nx;

        for (int j = 0; j < nx; ++j)
            norm[i * nx + j] = data[i * nx + j] - mean;
    }

    // 2. row-wise square-sum normalization

    // #pragma omp parallel for
    for (int i = 0; i < ny; ++i)
    {
        float sq_sum = 0;
        for (int j = 0; j < nx; ++j)
            // #pragma omp critical
            sq_sum += pow(norm[i * nx + j], 2);
        sq_sum = sqrt(sq_sum);
        for (int j = 0; j < nx; ++j)
            norm[i * nx + j] = norm[i * nx + j] / sq_sum;
    }

    // 3. upper-triangular matmul

    // 3.1. Allocate device memory
    size_t size_norm = ny * nx * sizeof(float);
    size_t size_result = ny * ny * sizeof(float);

    float *d_norm = nullptr, *d_r = nullptr;

    CHECK(hipMalloc(&d_norm, size_norm));
    CHECK(hipMalloc(&d_r, size_result));

    // 3.2. Move data from host -> device
    const int BLOCK_SIZE = 16;
    int grid_size = divup(ny, BLOCK_SIZE);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(grid_size, grid_size);

    hipMemcpy(d_norm, norm, size_norm, hipMemcpyHostToDevice);
    hipMemcpy(d_r, result, size_result, hipMemcpyHostToDevice);

    // 3.3. Execute kernel
    kernel<<<dimGrid, dimBlock>>>(d_norm, d_r, ny, nx);

    hipDeviceSynchronize();

    // 3.4. Move data from device back to host
    hipMemcpy(result, d_r, size_result, hipMemcpyDeviceToHost);

    CHECK(hipFree(d_r));
    CHECK(hipFree(d_norm));
}
